
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void pattern_blur_kernel(
    float *pattern, int pattern_size,
    unsigned char *in_img, int height, int width,
    unsigned char *out_img)
{
    int row_ind = threadIdx.x + blockIdx.x * blockDim.x;
    int col_ind = threadIdx.y + blockIdx.y * blockDim.y;

    if (row_ind >= height || col_ind >= width)
    {
        return;
    }

    int radius = (pattern_size - 1) / 2;

    for (int ch = 0; ch < 3; ++ch)
    {
        float new_value = 0.0;

        for (int di = -radius; di <= radius; ++di)
        {
            for (int dj = -radius; dj <= radius; ++dj)
            {
                int rind = row_ind + di;
                int cind = col_ind + dj;

                if (rind < 0) rind = 0;
                if (cind < 0) cind = 0;

                if (rind >= height) rind = height - 1;
                if (cind >= width) cind = width - 1;

                new_value += (
                    pattern[(di + radius) * pattern_size + (dj + radius)] *
                    in_img[(rind * width + cind) * 3 + ch]
                );
            }
        }

        if (new_value < 0) new_value = 0;
        if (new_value > 255) new_value = 255;

        out_img[(row_ind * width + col_ind) * 3 + ch] = (
            (unsigned char) new_value
        );
    }
}

int main(int argc, char *argv[])
{
    if (argc < 4)
    {
        printf(
            "Usage: %s input_pattern.txt input_img.bmp output.bmp\n", argv[0]);

        return 1;
    }

    FILE *input_pattern = fopen(argv[1], "r");

    int pattern_size;
    fscanf(input_pattern, "%d", &pattern_size);

    int pattern_sizeof = pattern_size * pattern_size * sizeof(float);
    float *h_pattern = (float *) malloc(pattern_sizeof);
    float pattern_sum = 0.0;

    for (int i = 0; i < pattern_size; ++i)
    {
        for (int j = 0; j < pattern_size; ++j)
        {
            float value;
            fscanf(input_pattern, "%f", &value);

            h_pattern[i * pattern_size + j] = value;
            pattern_sum += value;
        }
    }

    if (pattern_sum)
    {
        for (int i = 0; i < pattern_size; ++i)
        {
            for (int j = 0; j < pattern_size; ++j)
            {
                h_pattern[i * pattern_size + j] /= pattern_sum;
            }
        }
    }

    fclose(input_pattern);

    FILE *input_img = fopen(argv[2], "rb");

    int width, height;
    unsigned short int bpp;
    unsigned char header[138];

    fseek(input_img, 18, 0);
    fread(&width, sizeof(int), 1, input_img);

    fseek(input_img, 22, 0);
    fread(&height, sizeof(int), 1, input_img);

    fseek(input_img, 28, 0);
    fread(&bpp, sizeof(unsigned char), 1, input_img);

    fseek(input_img, 0, 0);
    fread(&header, sizeof(unsigned char), 138, input_img);

    int img_sizeof = height * width * 3 * sizeof(unsigned char);

    unsigned char *h_in_img = (unsigned char *) malloc(img_sizeof);

    unsigned int padding_size = (int)((width * bpp + 31) / 32) * 4 - width * 3;
    unsigned char *h_padding = (unsigned char *) malloc(padding_size);

    for (int i = 0; i < height; ++i)
    {
        for (int j = 0; j < width; ++j)
        {
            unsigned char b, g, r;

            fread(&b, sizeof(unsigned char), 1, input_img);
            fread(&g, sizeof(unsigned char), 1, input_img);
            fread(&r, sizeof(unsigned char), 1, input_img);

            h_in_img[(i * width + j) * 3] = r;
            h_in_img[(i * width + j) * 3 + 1] = g;
            h_in_img[(i * width + j) * 3 + 2] = b;
        }

        if (padding_size)
        {
            fread(&h_padding, padding_size, 1, input_img);
        }
    }

    fclose(input_img);

    unsigned char *h_out_img = (unsigned char *) malloc(img_sizeof);

    float *d_pattern;
    unsigned char *d_in_img;
    unsigned char *d_out_img;

    hipSetDevice(0);

    hipMalloc((void **) &d_pattern, pattern_sizeof);
    hipMalloc((void **) &d_in_img, img_sizeof);
    hipMalloc((void **) &d_out_img, img_sizeof);

    hipMemcpy(d_pattern, h_pattern, pattern_sizeof, hipMemcpyHostToDevice);
    hipMemcpy(d_in_img, h_in_img, img_sizeof, hipMemcpyHostToDevice);

    dim3 gridSize((int)(height / 16) + 1, int(width / 16) + 1);
    dim3 blockSize(16, 16);

    pattern_blur_kernel<<< gridSize, blockSize >>>(
        d_pattern, pattern_size,
        d_in_img, height, width,
        d_out_img);

    hipDeviceSynchronize();

    hipMemcpy(h_out_img, d_out_img, img_sizeof, hipMemcpyDeviceToHost);

    FILE *output_img = fopen(argv[3], "wb");

    fwrite(header, sizeof(unsigned char), 138, output_img);

    for (int i = 0; i < height; ++i)
    {
        for (int j = 0; j < width; ++j)
        {
            unsigned char r = h_out_img[(i * width + j) * 3];
            unsigned char g = h_out_img[(i * width + j) * 3 + 1];
            unsigned char b = h_out_img[(i * width + j) * 3 + 2];

            fwrite(&b, sizeof(unsigned char), 1, output_img);
            fwrite(&g, sizeof(unsigned char), 1, output_img);
            fwrite(&r, sizeof(unsigned char), 1, output_img);
        }

        if (padding_size)
        {
            fwrite(&h_padding, padding_size, 1, output_img);
        }
    }

    fflush(output_img);
    fclose(output_img);

    free(h_pattern);
    free(h_padding);
    free(h_in_img);
    free(h_out_img);

    hipFree(d_pattern);
    hipFree(d_in_img);
    hipFree(d_out_img);

    return 0;
}
